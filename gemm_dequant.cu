#include <iostream>
#include <vector>

#include "cutlass/cutlass.h"
#include "cutlass/arch/memory.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "gemm_with_dequant.h"
#include "gemm_ref.h"

void RunGemmDequant() {
  using ElementA = int8_t;
  using ElementB = int8_t;
  using ElementC = float;
  using ElementCompute = int32_t;
  using ElementD = ElementC;

  using LayoutA = cutlass::layout::RowMajor;
  using LayoutB = cutlass::layout::ColumnMajor;

  // using ThreadblockShape = cutlass::gemm::GemmShape<128, 128, 64>;
  // using WarpShape        = cutlass::gemm::GemmShape<64, 64, 64>;

  using ThreadblockShape = cutlass::gemm::GemmShape<64, 64, 64>;
  using WarpShape        = cutlass::gemm::GemmShape<32, 32, 64>;
  using InstructionShape = cutlass::gemm::GemmShape<16, 8, 32>;

  using OperatorClass = cutlass::arch::OpClassTensorOp;
  using ArchTag = cutlass::arch::Sm80;


  static int const kStages = 5;

  /// Linear scaling operator
  using EpilogueFunctorOp = cutlass::epilogue::thread::LinearCombination<
    ElementC,
    128 / cutlass::sizeof_bits<ElementC>::value,
    ElementCompute,
    ElementCompute
  >;

  using GemmDequant = cutlass::GemmDequant<
    ElementA, LayoutA,
    ElementB, LayoutB,
    ElementC,
    ElementCompute,
    OperatorClass,
    ArchTag,
    ThreadblockShape,
    WarpShape,
    InstructionShape,
    EpilogueFunctorOp,
    kStages
  >;

  using LayoutC = typename GemmDequant::LayoutC;

  // Initialize data

  int m = 40, k = 1024, n = 8192;

  int8_t* A_dev, *B_dev;
  float* C_dev = nullptr;
  float* D_dev;
  float* dequant_scale_dev;

  std::vector<int8_t> A(k * m, 1);
  // for (int i = 0; i < k; ++i) {
  //   for (int j = 0; j < m; ++j) {
  //     A[i * m + j] = int8_t(j+1);
  //   }
  // }
  for (int i = 0; i < m; ++i) {
    for (int j = 0; j < k; ++j) {
      A[i * k + j] = int8_t(i+1);
    }
  }
  

  std::vector<int8_t> B(n * k, 1);

  for (int i = 0; i < n; ++i) {
    for (int j = 0; j < k; ++j) {
      B[i * k + j] = int8_t(i) + j;
    }
  }
  std::vector<float> D(n * m);

  std::vector<float> dequant_scale(m, 1.0f);
  for (int i = 0; i < m; ++i) {
    dequant_scale[i] = i * 0.001f;
  }

  // Assume origin matrix is col-major

  hipMalloc(&A_dev, k * m); // k * m in row-major
  hipMalloc(&B_dev, n * k); // k * n in col-major
  hipMalloc(&D_dev, n * m * sizeof(float)); // m * n in row-major
  hipMalloc(&dequant_scale_dev, m * sizeof(float)); 

  hipMemcpy(A_dev, A.data(), k * m, hipMemcpyHostToDevice);
  hipMemcpy(B_dev, B.data(), n * k, hipMemcpyHostToDevice);
  hipMemcpy(dequant_scale_dev, dequant_scale.data(), m * sizeof(float), hipMemcpyHostToDevice);

  int64_t lda = LayoutA::packed({m, k}).stride(0);
  int64_t ldb = LayoutB::packed({k, n}).stride(0);
  int64_t ldc = LayoutC::packed({m, n}).stride(0);

  std::cout << "lda " << lda << "\n";
  std::cout << "ldb " << ldb << "\n";
  std::cout << "ldc " << ldc << "\n";

  cutlass::gemm::GemmCoord problem_size(m, n, k);

  GemmDequant::Arguments args(
      problem_size,
      {A_dev, lda},
      {B_dev, ldb},
      {C_dev, ldc},
      {D_dev, ldc},
      {dequant_scale_dev, 0},
      {
        ElementCompute(1.0f),
        ElementCompute(0.0f)
      }
    );

    //
    // Launch
    //

    GemmDequant gemm;

    // Initialize
    auto status = gemm.initialize(args);
    if (status != cutlass::Status::kSuccess) {
      std::cout  << "status " <<  int(status) << "\n";
      return;
    }

    // Run
    status = gemm();

    std::cout  << "status " <<  int(status) << "\n";
    hipDeviceSynchronize();

    hipMemcpy(D.data(), D_dev, n * m * sizeof(float), hipMemcpyDeviceToHost);

    // for (int i = 0; i < m; ++i) {
    //   for (int j = 0; j < n; ++j) {
    //     std::cout << D[i * n + j] << " ";
    //   }
    //   std::cout << "\n";
    // }

    // Ref
    // std::vector<int32_t> C_ref(m * n);
    // std::vector<float> D_ref(m * n);
    // GEMM<int8_t, int32_t, true>(A, B, C_ref, m, k, n);

    // for (int i = 0; i < m; ++i) {
    //   for (int j = 0; j < n; ++j) {
    //     D_ref[i * n + j] = float(C_ref[i * n + j]) * dequant_scale[i];
    //     if (D_ref[i * n + j] != D[i * n + j]) {
    //       std::cout << "Test fail in m " << i << " n " << j << "\n";
    //       std::cout << "Ring value is " << D_ref[i * n + j] << " but received " << D[i * n + j]   << "\n";
    //     }
    //   }
    // }

    // std::cout << "\n";
    // for (int i = 0; i < m; ++i) {
    //   for (int j = 0; j < n; ++j) {
    //     std::cout << D_ref[i * n + j] << " ";
    //   }
    //   std::cout << "\n";
    // }
    // return;


}

int main() {
  RunGemmDequant();
}